// FIXME: I don't know if we should do it this way, but there should
// only be one def of this. Should this be in the non-generic dir
// instead, along with other unique defs?
#ifndef THC_TENSOR_FILL_GENERIC_DEFS
#define THC_TENSOR_FILL_GENERIC_DEFS

template <typename T>
struct TensorFillOp {
  TensorFillOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* v) { *v = val; }

  const T val;
};

// FIXME: hack to deal with `half`
template <typename IN, typename OUT>
struct ValueConverter {
  static OUT get(IN v) { return v; }
};

template <>
struct ValueConverter<half, short> {
  static short get(half v) { return v.x; }
};

#endif

#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorFill.cu"
#else

THC_API void
THCTensor_(fill)(THCState* state, THCTensor *self_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));

  typename TensorUtils<THCTensor>::DataType v =
    ValueConverter<real,
                   typename TensorUtils<THCTensor>::DataType>::get(value);

  if (!cutorch_pointwiseApply1(
        state, self_,
        // FIXME: `real` should be sufficient, but code can't handle
        // half at the moment. In any case, this is more C++-ish
        TensorFillOp<typename TensorUtils<THCTensor>::DataType>(v))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

THC_API void
THCTensor_(zero)(THCState *state, THCTensor *self_)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));
  if (THCTensor_(isContiguous)(state, self_)) {
    THCudaCheck(hipMemsetAsync(THCTensor_(data)(state, self_),
                                0,
                                sizeof(real) * THCTensor_(nElement)(state, self_),
                                THCState_getCurrentStream(state)));
  } else {
    if (!cutorch_pointwiseApply1(
          state, self_,
          // FIXME: `real` should be sufficient, but code can't handle
          // half at the moment. In any case, this is more C++-ish
          TensorFillOp<typename TensorUtils<THCTensor>::DataType>(
            (typename TensorUtils<THCTensor>::DataType) 0))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

#endif
